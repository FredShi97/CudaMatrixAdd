#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


#define BLOCK_SIZE_ROW 16
#define BLOCK_SIZE_COL 16

double getTimeStamp() {
    struct timeval  tv ; gettimeofday( &tv, NULL ) ;
    return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}


__global__ void f_siggen(float *d_x, float *d_y, float *d_z, int row_size, int col_size){
    
    //add one row above and one row below to accommendate for row - 1 and row + 1 read
    __shared__ float x_shared[(BLOCK_SIZE_ROW + 2) * BLOCK_SIZE_COL]; 
    
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int offset; 
    int shared_index;

    if (x < col_size && y < row_size){
        offset = y * col_size + x;  
        shared_index = (threadIdx.y + 1) * blockDim.x + threadIdx.x; 
        //copy out d_x based on block index
        x_shared[shared_index] = d_x[offset]; 
        //if its first row, copy out one row above from d_x
        if (threadIdx.y == 0 && (y - 1 >= 0))
            x_shared[threadIdx.x] = d_x[(y - 1) * col_size + x]; 
        //if its last row, copy out one row below from d_x
        if ((threadIdx.y == blockDim.y - 1) && (y + 1 < row_size))
            x_shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x] = d_x[(y + 1) * col_size + x];
    }
    
     __syncthreads(); 

    if (x < col_size && y < row_size){
        float output = x_shared[shared_index] - d_y[offset]; 

        //coalesced access. 
        if (x - 1 >= 0) 
            output -= d_y[y * col_size + x - 1];
        if (x - 2 >= 0)
            output -= d_y[y * col_size + x - 2]; 

        // read from shared memory 
        if (y - 1 >= 0)
            output += x_shared[threadIdx.y * blockDim.x + threadIdx.x];
        if (y + 1 < row_size)
            output +=  x_shared[(threadIdx.y + 2) * blockDim.x + threadIdx.x];

        d_z[offset] = output; 
    }
    

    
}

int main(int argc, char **argv) {

    if (argc != 3){
        printf("Program takes 3 args, <row_size> <col_size>"); 
        return 0; 
    }

    int row_size = atoi(argv[1]);
    int col_size = atoi(argv[2]); 

    float *h_x, *h_y, *h_z, *d_x, *d_y, *d_z; 
    int n_size = row_size * col_size;

    hipHostAlloc((void**) &h_x, n_size * sizeof(float), 0); 
    hipHostAlloc((void**) &h_y, n_size * sizeof(float), 0); 
    hipHostAlloc((void**) &h_z, n_size * sizeof(float), 0); 
    hipMalloc((void**) &d_x, n_size * sizeof(float));
    hipMalloc((void**) &d_y, n_size * sizeof(float));
    hipMalloc((void**) &d_z, n_size * sizeof(float));

    for (int i = 0; i < row_size; i++){
        for (int j = 0; j < col_size; j++){
            int offset = i * col_size + j; 
            h_x[offset] = (float) ((i+j) % 100) / 2.0; 
            h_y[offset] = (float) 3.25 * ((i+j) % 100); 
        }
    }



    dim3 blockSize(BLOCK_SIZE_COL, BLOCK_SIZE_ROW); 
    int gridX = (col_size + blockSize.x - 1) / blockSize.x;
    int gridY = (row_size + blockSize.y - 1) / blockSize.y;
    dim3 gridSize(gridX, gridY); 


    double startTime = getTimeStamp(); 
    double endTime; 
    double totalStartTime = startTime; 

    hipMemcpy(d_x, h_x, n_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n_size * sizeof(float), hipMemcpyHostToDevice);

    endTime = getTimeStamp();
    double CPU_GPU_Transfer_time = endTime - startTime; 
    startTime = endTime;

    f_siggen<<<gridSize,blockSize>>>(d_x, d_y, d_z, row_size, col_size); 
    hipDeviceSynchronize(); 
    endTime = getTimeStamp();
    double kernel_time = endTime - startTime; 
    startTime = endTime;

    hipMemcpy(h_z, d_z, n_size * sizeof(float), hipMemcpyDeviceToHost);
    endTime = getTimeStamp();
    double GPU_CPU_Transfer_time = endTime - startTime; 

    double total_time = endTime - totalStartTime; 


    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z); 


    printf("%f %f %f %f %f \n", total_time, CPU_GPU_Transfer_time, kernel_time, 
    GPU_CPU_Transfer_time, h_z[5*col_size + 5]);

    
    for (int i = 0; i < row_size; i++){
        for (int j = 0; j < col_size; j++){
            int offset = i * col_size + j; 
            float out = 0.0;
            out = h_x[offset] - h_y[offset];
            if (i - 1 >= 0)
                out += h_x[(i - 1) * col_size + j];
            if (i + 1 < row_size)
                out += h_x[(i + 1) * col_size + j];
            if (j - 1 >= 0)
                out -= h_y[i * col_size + j - 1];
            if (j - 2 >= 0)
                out -= h_y[i * col_size + j - 2];   

            //printf("X is %f, Y is %f, row %d, col %d \n", h_x[offset], h_y[offset], i, j);
            
            if (h_z[offset] != out)
                printf("CPU calculated is %f, GPU is %f, row %d, col %d \n", out, h_z[offset], i, j);
        }
    }

    hipHostFree(h_x);
    hipHostFree(h_y);
    hipHostFree(h_z);
  

    return 0;
}

